
#include <hip/hip_runtime.h>
#include "hip/hip_complex.h"

template<size_t NBEAMS, size_t NANTS, size_t NCHANS, size_t NTIME, size_t NPOLS, size_t TBLOCK>
__global__ void beamformer(const char2* in, const hipFloatComplex* phasor, hipFloatComplex* out) {
    int ti = threadIdx.x + (blockIdx.y * TBLOCK);
    int ch = blockIdx.x;
    int bi = threadIdx.x;

    // Load the phasors to shared memory.
    __shared__ hipFloatComplex phr_cache[NBEAMS][NANTS][NPOLS];

    int iy = (ch * NPOLS) + (bi * NPOLS * NCHANS * NANTS);
    int dy = NPOLS * NCHANS;

    if (threadIdx.x < NBEAMS) {
        for (int a = 0; a < NANTS; a++, iy += dy) {
            phr_cache[threadIdx.x][a][0] = phasor[iy+0];
            phr_cache[threadIdx.x][a][1] = phasor[iy+1];
        }
    }

    __syncthreads();

    // Load the antenna values to registers.
    hipFloatComplex ant_cache[NANTS][NPOLS];

    int ix = (ch * NTIME) + (ti);
    int dx = NTIME * NCHANS;

    for (int a = 0; a < NANTS; a++, ix += dx) {
        const char4 tmp = reinterpret_cast<const char4*>(in)[ix];
        ant_cache[a][0] = make_hipFloatComplex(tmp.x, tmp.y);
        ant_cache[a][1] = make_hipFloatComplex(tmp.z, tmp.w);
    }

    // Multiply and accumulate.
    int iz = (ch * NTIME) + ti;
    int dz = NTIME * NCHANS;

    for (int b = 0; b < NBEAMS; b++, iz += dz) {
        hipFloatComplex acc[NPOLS] = {{0.0, 0.0}};

        for (int a = 0; a < NANTS; a++) {
            acc[0] = hipCaddf(acc[0], hipCmulf(ant_cache[a][0], phr_cache[b][a][0]));
            acc[1] = hipCaddf(acc[1], hipCmulf(ant_cache[a][1], phr_cache[b][a][1]));
        }

        reinterpret_cast<float4*>(out)[iz] = *reinterpret_cast<float4*>(acc);
    }
}
