
#include <hip/hip_runtime.h>
template<typename T, size_t N>
__global__ void checker(T a, T b, unsigned long long int* result) {
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = threadID; i < N; i += numThreads) {
#ifdef DEBUG
        if (threadIdx.x == 0 && blockIdx.x == 0) {
            for (int i = 0; i < N; i++) {
                const char* status = (abs(a[i], b[i]) > 0.01) ? "NOK" : " OK";
                printf("[%04d] %s %f != %f\n", i, status, (T)a[i], (T)b[i]);
            }
        }
#endif
        if (abs((double)a[i] - (double)b[i]) > 0.01) {
            atomicAdd(result, 1);
        }
    }
}
