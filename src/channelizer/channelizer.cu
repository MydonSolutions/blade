
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

// 4-point FFT

template<size_t N, size_t NFFT, size_t NPOLS>
__global__ void fft_4pnt(const hipFloatComplex* input, hipFloatComplex* output) {
    const int numThreads = (blockDim.x * gridDim.x) * (NFFT * NPOLS);
    const int threadID = (blockIdx.x * blockDim.x + threadIdx.x) * (NFFT * NPOLS);

    for (int i = threadID; i < N; i += numThreads) {
        for (int j = i; j < i + NPOLS; j += 1) {
            // TODO: Fix indexes for NPOLS.
            // TODO: Add reordering index.

            const float2 a = input[j+0];
            const float2 b = input[j+2];
            const float2 c = input[j+4];
            const float2 d = input[j+6];

            const float r1 = a.x - c.x;
            const float r2 = a.y - c.y;
            const float r3 = b.x - d.x;
            const float r4 = b.y - d.y;

            const float t1 = a.x + c.x;
            const float t2 = a.y + c.y;
            const float t3 = b.x + d.x;
            const float t4 = b.y + d.y;

            const float a3 = t1 - t3;
            const float a4 = t2 - t4;
            const float b3 = r1 - r4;
            const float b2 = r2 - r3;

            const float a1 = t1 + t3;
            const float a2 = t2 + t4;
            const float b1 = r1 + r4;
            const float b4 = r2 + r3;

            output[j+0] = make_hipFloatComplex(a1, a2);
            output[j+2] = make_hipFloatComplex(b1, b2);
            output[j+4] = make_hipFloatComplex(a3, a4);
            output[j+6] = make_hipFloatComplex(b3, b4);
        }
    }
}
